#include "hip/hip_runtime.h"
#include "CudaTest.cuh"
#include <stdio.h>

CudaTest::CudaTest(void) {

}

CudaTest::~CudaTest() {

}

__global__ void sum_kernel(int a, int b, int* c) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a + b;
}
__global__ void multiply_kernel(int a, int b, int* c) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a * b;
}

int CudaTest::sum_cuda(int a, int b, int* c) {
	int* f;
	hipMalloc((void**)&f, sizeof(int) * 1);
	hipMemcpy(f, c, sizeof(int) * 1, hipMemcpyDeviceToDevice);

	sum_kernel << <1, 1 >> > (a, b, f);
	hipMemcpy(c, f, sizeof(int) * 1, hipMemcpyDeviceToHost);

	hipFree(f);

	return true;
}

int CudaTest::multiply_cuda(int a, int b, int* c) {
	int* g;
	hipMalloc((void**)&g, sizeof(int) * 1);
	hipMemcpy(g, c, sizeof(int) * 1, hipMemcpyDeviceToDevice);

	multiply_kernel << <1, 1 >> > (a, b, g);
	hipMemcpy(c, g, sizeof(int) * 1, hipMemcpyDeviceToHost);

	hipFree(g);

	return true;
}